#include "hip/hip_runtime.h"
#include "FitnessCUDA.h"
#include <algorithm>
#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

using namespace geneticalgorithm;

__global__ void deviceCode(Population &population, Parameters params) {
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	population[i].setFitnessInfo(fitness::evaluateCUDA(population[i], params));
}

__host__ void geneticalgorithm::evaluateWithCUDA(Population &population, Parameters params) {
	int numElements = population.chromosomes().size();
	Population devPop;
	hipMemcpy(&devPop, &population, sizeof(Population), hipMemcpyHostToDevice);
	deviceCode<<<1, numElements>>> (devPop, params);
	hipMemcpy(&population, &devPop, sizeof(Population), hipMemcpyDeviceToHost);
	hipFree(&devPop);
}